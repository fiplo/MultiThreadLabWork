#include "hip/hip_runtime.h"
#include <string>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include ""
#include <algorithm>
#include <fstream>

using namespace std;

const int ARRAY_SIZE = 50;
const int CHAR_ARRAY_SIZE = 11;
const int STRING_SIZE = 260;
struct User {
    string Name;
    int Age;
    double Balance;
};


void read(Item users[], string fileName, int &count) {
	ifstream  ifs(fileName);
	string Name;
	int Age;
	double Balance;
  int i = 0;

	if (ifs.fail()) {
		cout << "Error opening file" + fileName << endl;
		exit(1);
	}
	while(!ifs.fail()){
		ifs >> Name >> Age >> Balance;
    users[i].Name = Name;
		users[i].Age = Age;
		users[i].Balance = Balance;
    if( ifs.eof() ) break;
    i++;
	}
  count = i;

}
__global__ void addition(User* input, User* output, size_t n, int step)
{
	int index = threadIdx.x;
	for (int i = index; i < n; i += step) {
		output[index].Age += input[i].Age;
		output[index].Balance += input[i].Balance;
    output[index].Name += input[i].Name;
	}
}
int main() {

  string fileName = "../data/Paulius_Ratkevicius_L1_dat_1_ResPlain.txt";
  regex outputdir(".txt");
  string outputFile = regex_replace(fileName, outputdir, "Res.txt");

  int counter;

	int threads = 4;
	User input[ARRAY_SIZE];
	read(input, fileName, counter);
	User* hostData = input;
	User* deviceData;
	User* hostA = new User[counter];
	User* deviceA;// = new User[ARRAY_SIZE];
	hipMalloc(&deviceData, counter * sizeof(User));
	hipMalloc(&deviceA, counter * sizeof(User));
	hipMemcpy(deviceData, input, counter * sizeof(User), hipMemcpyHostToDevice);
	addition << <1, threads >> > (deviceData, deviceA, counter, threads);
	//hipMemcpy(hostData, deviceData, ARRAY_SIZE * sizeof(User), hipMemcpyDeviceToHost);
	hipMemcpy(hostA, deviceA, counter * sizeof(User), hipMemcpyDeviceToHost);
	for (int i = 0; i < threads; i++) {
		cout << hostA[i].Name << " " << hostA[i].Age << " " << hostA[i].Balance << "\n";
	}
	return 0;
}

