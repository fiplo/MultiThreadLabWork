#include "hip/hip_runtime.h"
#include <string>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include ""
#include <algorithm>
#include <fstream>
#include "json.hpp"

using namespace std;
using json = nlohmann::json;

const int ARRAY_SIZE = 27;//25;
const int CHAR_ARRAY_SIZE = 11;
const int STRING_SIZE = 260;

struct Item {
	char stringas[CHAR_ARRAY_SIZE];
	int intas;
	double doublas;
};
struct Answer {
	char stringas[STRING_SIZE];
	int intas;
	double doublas;
};

namespace ns{
    struct User {
        string Name;
        int Age;
        double Balance;
    };


    void to_json(json& j, const User& p){
        j = json{{"name", p.Name}, {"age", p.Age}, {"balance", p.Balance}};
    }

    void from_json(const json& j, User& p){
        j.at("name").get_to(p.Name);
        j.at("age").get_to(p.Age);
        j.at("balance").get_to(p.Balance);
    }

}

ns:User* ParseJson(string fileName){
    ifstream fileStream(fileName);
    if (fileStream.fail()) {
        cout << "Error while opening stream of input file: " << fileName << endl;
        exit(1);
    }
    ns:User users[ARRAY_SIZE]
    int counter = 0;
    json j = json::parse(fileStream);
    for (auto it: j.items() )
    {
        json &user = it.value();
        users[counter] = user.get<ns::User>();
    }
    return users;
}
void OutputJson(string fileName, ns:User users[]){
    json output;
    int i = 0;
    while(users[i].Age != 0 && i < ARRAY_SIZE){
        output.push_back(users[i]);
    }
    ofstream results(fileName);
    results << output.dump(1, '\t') << endl;
}


void read(Item items[]) {
	//ifstream  ifs("IFK-7_NojusD_L3_dat.txt");
	ifstream  ifs("IFK_NojusD_L3_dat.txt");
	string stringas;
	int intas;
	double doublas;

	if (ifs.fail()) {
		cout << "Error opening file (IFK-7_NojusD_L3_dat.txt)" << endl;
		exit(1);
	}
	for(size_t i = 0; i <ARRAY_SIZE; i++){
		ifs >> stringas >> intas >> doublas;
		for (int j = 0; j < CHAR_ARRAY_SIZE;j++) {
			if (stringas[j] == 0) {
				items[i].stringas[j] = 0;
				break;
			}
			items[i].stringas[j] = (char)stringas[j];
		}
		items[i].intas = intas;
		items[i].doublas = doublas;
	}

}
__global__ void addition(Item* data, Answer* sum, size_t n, int step)
{
	int index = threadIdx.x;
	int iterator = 0;
	for (int i = index; i < n; i += step) {
		sum[index].intas += data[i].intas;
		sum[index].doublas += data[i].doublas;
		for (int j = 0; j < CHAR_ARRAY_SIZE; j++) {
			if (data[i].stringas[j] == 0) {
				break;
			}
			sum[index].stringas[iterator] = data[i].stringas[j];
			//printf("%c", sum[thread_id].stringas[iterator]);
			iterator++;
		}
	}
}
int main() {
	int threads = 4;
	Item items[ARRAY_SIZE];
	read(items);
	Item* hostData = items;
	Item* deviceData;
	Answer* hostA = new Answer[ARRAY_SIZE];
	Answer* deviceA;// = new Answer[ARRAY_SIZE];
	hipMalloc(&deviceData, ARRAY_SIZE * sizeof(Item));
	hipMalloc(&deviceA, ARRAY_SIZE * sizeof(Answer));
	hipMemcpy(deviceData, items, ARRAY_SIZE * sizeof(Item), hipMemcpyHostToDevice);
	addition << <1, threads >> > (deviceData, deviceA, ARRAY_SIZE, threads);
	//hipMemcpy(hostData, deviceData, ARRAY_SIZE * sizeof(Item), hipMemcpyDeviceToHost);
	hipMemcpy(hostA, deviceA, ARRAY_SIZE * sizeof(Answer), hipMemcpyDeviceToHost);
	for (int i = 0; i < threads; i++) {
		cout << hostA[i].intas << " " << hostA[i].doublas << " " << hostA[i].stringas << "\n";
	}
	return 0;
}

